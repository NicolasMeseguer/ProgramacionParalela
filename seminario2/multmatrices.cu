#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define N 8 //Filas
#define M 8 //Columnas

__global__ void multMatrices(float *c, float *a, float *b){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  int ix = (blockIdx.y*blockDim.y+threadIdx.y)*N+(blockIdx.x*blockDim.x+threadIdx.x);
  if(ix<N*M) {
    int adder = 0;
    for(int i=0;i<N;++i)
      adder+= (float) a[(blockIdx.y*blockDim.y+threadIdx.y)*N+(i)]*b[(i)*N+(blockIdx.x*blockDim.x+threadIdx.x)];
    c[ix]=adder;
  }
}

int main() {
	int memsize = sizeof(float )*N*M;
  float *h_a,*h_b,*h_c; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);
	h_c=(float *)malloc(memsize);

	for(int i=0; i<N*M; ++i)
		h_a[i]=h_b[i]=h_c[i]=(float) 1.0f;

  float *d_a,*d_b,*d_c; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
  hipMalloc(&d_c, memsize);

  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, memsize, hipMemcpyHostToDevice);
 
  dim3 block(4,4);
  dim3 thread(2,2); 
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block.x, thread.x);
  multMatrices <<<block,thread>>> (d_c, d_a, d_b);//El multiplicar ambos numeros tiene que darme N

  hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);

	printf("Resultado multiplicacion de matrices: \n");
	for(int i=0; i<N*M; ++i){
		printf("%f, ", h_c[i]);
    if(i!=0 && i%N==(N-1))
      printf("\n");
  }

	printf("\n");

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

	return 0;
}
