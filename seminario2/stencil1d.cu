#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define TAM 10
#define TAMBLOCK 2

__global__ void sumaVectores(float *b, float *a){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  	int i = blockIdx.x*blockDim.x+threadIdx.x; //Obtengo el indice para cada iteracion de la funcion sobre cada hilo
    if(i<TAM && i!=0 && i!=TAM-1)
		  b[i]=(a[i-1]*a[i]*a[i+1])/3;
}

int main() {
	int memsize = sizeof(float)*TAM;
	
  float *h_a,*h_b; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);

  float *d_a,*d_b; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);

	for(int i=0; i<TAM; ++i)
		h_a[i]=(float)(rand()%50);

  //Envio el contenido de los arrays(h_a, h_b) CONTENIDO ! Al espacio de memoria ya reservado en la GPU(d_a, d_b). CPU -> GPU | Host -> Device
  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_a, memsize, hipMemcpyHostToDevice);
    
  int block = ceilf((float)TAM/TAMBLOCK);
  int thread = TAMBLOCK;
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block, thread);
  sumaVectores <<<block,thread>>> (d_b, d_a);//El multiplicar ambos numeros tiene que darme N

  //Envio el contenido del array(d_c) CONTENIDO ! Al espacio de memoria ya reservado en la CPU(h_c). GPU -> CPU | Device -> Host
  hipMemcpy(h_b, d_b, memsize, hipMemcpyDeviceToHost);

	printf("Resultado del tercer vector, c: \n");
	for(int i=0; i<TAM; ++i)
		printf("%f, ", h_b[i]);

	printf("\n");

  free(h_a);
  free(h_b);
  hipFree(d_a);
  hipFree(d_b);
	return 0;
}
