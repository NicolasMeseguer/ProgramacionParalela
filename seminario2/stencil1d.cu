#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define TAM 8
#define NITERACIONES 40
#define TAMBLOCK 2

__global__ void stencil(float *a, float *b){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  int i = blockIdx.x*blockDim.x+threadIdx.x+1;
  b[i]=(a[i-1]+a[i]+a[i+1])/3;
}

int main() {
	int memsize = sizeof(float)*TAM;
  float *h_a,*h_b; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);

  for(int i=0;i<TAM;++i){
    h_a[i]=h_b[i]=(float)(rand()%20);
  }

  float *d_a,*d_b; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);

  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
 
  dim3 block((TAM-2)/TAMBLOCK);
  dim3 thread(TAMBLOCK); 
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block.x, thread.x);
  for(int j=0;j<NITERACIONES;++j){
    stencil <<<block,thread>>> (d_a, d_b);
    hipDeviceSynchronize();
    float *aux=d_b;
    d_b=d_a;
    d_a=aux;
  }

  hipMemcpy(h_a, d_a, memsize, hipMemcpyDeviceToHost);
  hipMemcpy(h_b, d_b, memsize, hipMemcpyDeviceToHost);

  printf("Vector A:\n");
  for(int i=0;i<TAM;++i)
    printf("%f, ", *(h_a+i));
  
  printf("\n\nVector B:\n");
  for(int i=0;i<TAM;++i)
    printf("%f, ", *(h_b+i));
  printf("\n");

  free(h_a);
  free(h_b);
  hipFree(d_a);
  hipFree(d_b);
	return 0;
}
