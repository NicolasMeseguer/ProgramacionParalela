#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define N 8

__global__ void reduceVector (float *d_a) {


  int i = blockIdx.x*blockDim.x+threadIdx.x; 

  d_a[i] = d_a[i] + d_a[i+(N/2)];
  __syncthreads();
  if (threadIdx.x<(blockDim.x/2))
    d_a[i] = d_a[i] +d_a[i+(N/4)];
  __syncthreads(); 
  if (threadIdx.x<(blockDim.x/4))
    d_a[i] = d_a[i] +d_a[i+(N/8)];
}

int main () {

	int memsize = N*sizeof(float);
	float *a = (float *) malloc (memsize);  
	float resultado =0.0f; 
  float *d_a; 
  hipMalloc (&d_a, memsize);
 

	for (int i=0;i<N; ++i){
		a[i]=1.0f;
	}

  hipMemcpy (d_a, a, memsize, hipMemcpyHostToDevice);
	dim3 block (1);
  dim3 thread (N/2);

  reduceVector<<< block, thread>>> (d_a); 

  hipMemcpy (&resultado,d_a, sizeof(float), hipMemcpyDeviceToHost);

	printf ("%f",resultado);

	
	printf ("\n");
}






