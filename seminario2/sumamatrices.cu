#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define N 4 //Filas
#define M 4 //Columnas

__global__ void sumaMatrices(float *c, float *a, float *b){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  int i = (blockIdx.y*blockDim.y+threadIdx.y)*N+(blockIdx.x*blockDim.x+threadIdx.x);
  c[i]=a[i]+b[i];
}

int main() {
	int memsize = sizeof(float )*N*M;
  float *h_a,*h_b,*h_c; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);
	h_c=(float *)malloc(memsize);

  

  float *d_a,*d_b,*d_c; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
  hipMalloc(&d_c, memsize);

	for(int i=0; i<N*M; ++i)
		h_a[i]=h_b[i]=(float) i;

  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
 
  dim3 block(2,2);
  dim3 thread(2,2); 
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block.x, thread.x);
  sumaMatrices <<<block,thread>>> (d_c, d_a, d_b);//El multiplicar ambos numeros tiene que darme N

  //Envio el contenido del array(d_c) CONTENIDO ! Al espacio de memoria ya reservado en la CPU(h_c). GPU -> CPU | Device -> Host
  hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);

	printf("Resultado del tercer vector, c: \n");
	for(int i=0; i<N*M; ++i){
		printf("%f, ", h_c[i]);
    if(i%N==0)
      printf("\n");
  }

	printf("\n");

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

	return 0;
}
