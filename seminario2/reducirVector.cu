#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define TAM 10
#define TAMBLOCK 2

__global__ void reducirVector(float *dest, float *origin, int *BLOCKS){
  if(blockIdx.x==0){
    float counter=0.0f;
    for(int i=0;i<*BLOCKS;++i){
        for(int j=0;j<TAMBLOCK;++j){
          counter+=origin[i*blockDim.x+j];
      }
    }
  *dest=counter;
  }
}

void inicializarVector(float *vector){
  for(int i=0;i<TAM;++i)
    vector[i]=(float)(rand()%100);
}

void mostrarVector(float *vector){
  for(int i=0;i<TAM;++i)
    printf("%f, ",*(vector+i));
  printf("\n");
}

int main() {
  int memsize = sizeof(float)*TAM;
  float *h_vector, *h_aux;
  h_vector = (float *) malloc(memsize);
  h_aux    = (float *) malloc(memsize);
    
  inicializarVector(h_vector);
  mostrarVector(h_vector);

  float *d_vector, *d_aux;
  int *d_block;
  hipMalloc(&d_vector, memsize);
  hipMalloc(&d_aux, memsize);
  hipMalloc(&d_block, sizeof(int));

  hipMemcpy(d_vector, h_vector, memsize, hipMemcpyHostToDevice);

  int block = ceilf((float)TAM/TAMBLOCK);
  int thread = TAMBLOCK;
  hipMemcpy(d_block, &block, sizeof(int), hipMemcpyHostToDevice);
  reducirVector <<<block,thread>>> (d_aux, d_vector, d_block);

  hipMemcpy(h_aux, d_aux, memsize, hipMemcpyDeviceToHost);
  printf("Vector reducido, valor: %f\n", *(h_aux));
 
  free(h_vector);
  free(h_aux);
  hipFree(d_vector);
  hipFree(d_aux);
  return 0;
}
