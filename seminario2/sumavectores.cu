#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define TAM 8
#define TAMBLOCK 2

__global__ void sumaVectores(float *c, float *a, float *b){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  	int ix = blockIdx.x*blockDim.x+threadIdx.x; //Obtengo el indice para cada iteracion de la funcion sobre cada hilo
    if(ix<(int)ceilf(TAM/2)) {
		  c[ix]=a[ix]+b[ix];
      c[ix+(int) ceilf(TAM/2)] = a[ix+(int) ceilf(TAM/2)] + b[ix+(int) ceilf(TAM/2)];
    }
    /*for(int i=ix;i<TAM;i+=(int)ceilf(TAM/2)){
      c[i]=a[i]+b[i];
    }*/
}

int main() {
	int memsize = sizeof(float)*TAM;
	
  float *h_a,*h_b,*h_c; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);
	h_c=(float *)malloc(memsize);

  float *d_a,*d_b,*d_c; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
  hipMalloc(&d_c, memsize);

	for(int i=0; i<TAM; ++i)
		h_a[i]=h_b[i]=1.0f;

  /*ESTO NO FUNCIONA, UNA INSTRUCCION NO PUEDE ACCEDER A UN AREA DE MEMORIA DE UN PERIFERICO
  for(int i=0; i<TAM; ++i){
    *(d_a+i)=*(h_a+i);
    *(d_b+i)=*(h_b+i);
    *(d_c+i)=*(h_c+i);
  }
  */
  //Envio el contenido de los arrays(h_a, h_b) CONTENIDO ! Al espacio de memoria ya reservado en la GPU(d_a, d_b). CPU -> GPU | Host -> Device
  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, memsize, hipMemcpyHostToDevice);//No haria falta puesto que h_c esta vacio pero bueno...
  /**/
  
  int block = ceilf(((float)TAM/TAMBLOCK)/2);
  int thread = TAMBLOCK;
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block, thread);
  sumaVectores <<<block,thread>>> (d_c, d_a, d_b);//El multiplicar ambos numeros tiene que darme N

  //Envio el contenido del array(d_c) CONTENIDO ! Al espacio de memoria ya reservado en la CPU(h_c). GPU -> CPU | Device -> Host
  hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);

	printf("Resultado del tercer vector, c: \n");
	for(int i=0; i<TAM; ++i)
		printf("%f, ", h_c[i]);

	printf("\n");

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

	return 0;
}
