#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define TAM 8

__global__ void sumaVectores(float *c, float *a, float *b){ //Kernel, salto a la GPU.
  	int i = blockIdx.x threadIdx.x

		c[i]=a[i]+b[i];
}

int main() {
	int memsize = sizeof(float)*TAM;
	
  float *h_a,*h_b,*h_c; //Arrays en el host (CPU & RAM)
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);
	h_c=(float *)malloc(memsize);

  float *d_a,*d_b,*d_c; //Arrays en la GPU
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
  hipMalloc(&d_c, memsize);

	for(int i=0; i<TAM; ++i)
		h_a[i]=h_b[i]=1.0f;

  /*ESTO NO FUNCIONA, UNA INSTRUCCION NO PUEDE ACCEDER A UN AREA DE MEMORIA DE UN PERIFERICO
  for(int i=0; i<TAM; ++i){
    *(d_a+i)=*(h_a+i);
    *(d_b+i)=*(h_b+i);
    *(d_c+i)=*(h_c+i);
  }
  */
  //Envio el contenido de los arrays(h_a, h_b) CONTENIDO ! Al espacio de memoria ya reservado en la GPU(d_a, d_b). CPU -> GPU | Host -> Device
  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, memsize, hipMemcpyHostToDevice);//No haria falta puesto que h_c esta vacio pero bueno...
  /**/
  
  sumaVectores <<< 8, 1>>> (d_c, d_a, d_b);

  //Envio el contenido del array(d_c) CONTENIDO ! Al espacio de memoria ya reservado en la CPU(h_c). GPU -> CPU | Device -> Host
  hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);

	printf("Resultado del tercer vector, c: \n");
	for(int i=0; i<TAM; ++i)
		printf("%f, ", h_c[i]);

	printf("\n");

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

	return 0;
}
