#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <math.h>

#define TAM 7
#define TAMBLOCK 4

__global__ void sumaVectores(float *c, float *a, float *b, int *escalar){ //Kernel, salto a la GPU. Esta funcion es ejecutada por todos los hilos al mismo tiempo.
  	int i = threadIdx.x; //Obtengo el indice para cada iteracion de la funcion sobre cada hilo
    for(;i < TAM; i+=TAMBLOCK)
		  c[i]=(*(escalar)*a[i])+b[i];
}

void inicializarArrays(float *a, float *b){
	for(int i=0; i<TAM; ++i)
		a[i]=b[i]=1.0f;//(float)(rand()%50);
}

int main() {
	int memsize = sizeof(float)*TAM;
	
  /*HOST SIDE CPU & RAM*/
  //Arrays
  float *h_a,*h_b,*h_c;
	h_a=(float *)malloc(memsize);
	h_b=(float *)malloc(memsize);
	h_c=(float *)malloc(memsize);
  //Escalares
  int h_escalar=30;
  int aux_escalar;
  /**/

  //Inicializar valores arrays.
  inicializarArrays(h_a, h_b);  
  
  /*DEVICE SIDE GPU & GRAM*/
  //Arrays
  float *d_a,*d_b,*d_c;
  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
  hipMalloc(&d_c, memsize);
  //Escalares
  int *d_escalar;
  hipMalloc((void**)&d_escalar, sizeof(int));

  //Envio el contenido de los arrays(h_a, h_b) CONTENIDO ! Al espacio de memoria ya reservado en la GPU(d_a, d_b). CPU -> GPU | Host -> Device
  hipMemcpy(d_a, h_a, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, memsize, hipMemcpyHostToDevice);//No haria falta puesto que h_c esta vacio pero bueno...
  hipMemcpy(d_escalar, &h_escalar, sizeof(int), hipMemcpyHostToDevice);

  /*Para comprobar que se ha copiado correctamente vamos a volver a traerlo a host y comprobar su valor mediante un printf.*/
  hipMemcpy(&aux_escalar, d_escalar, sizeof(int), hipMemcpyDeviceToHost); 

  int block = 1;
  int thread = TAMBLOCK;
  printf("El numero de bloques es %d, y el numero de hilos es %d\n", block, thread);
  printf("El valor del escalar en HOST: %d, en DEVICE: %d\n", h_escalar, aux_escalar);
  sumaVectores <<<block,thread>>> (d_c, d_a, d_b, d_escalar);//El multiplicar ambos numeros tiene que darme N

  //Envio el contenido del array(d_c) CONTENIDO ! Al espacio de memoria ya reservado en la CPU(h_c). GPU -> CPU | Device -> Host
  hipMemcpy(h_c, d_c, memsize, hipMemcpyDeviceToHost);

	printf("Resultado del tercer vector, c: \n");
	for(int i=0; i<TAM; ++i)
		printf("%f, ", h_c[i]);

	printf("\n");

  free(h_a);
  free(h_b);
  free(h_c);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

	return 0;
}
