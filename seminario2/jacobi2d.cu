#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 8
#define TAMBLOCK 2

#define LIMIT 10

__global__ void jacobi2d(float *A, float *B){
    int ind = (((blockIdx.y*blockDim.y+threadIdx.y)+1)*N)+((blockIdx.x*blockDim.x+threadIdx.x)+1);
		*(B+ind)=*(A+ind)+*(A+ind+1)+*(A+ind-1)+*(A+ind+N)+*(A+ind-N);
}

int main(){
  int memsize=N*N*sizeof(float);
	float *A = (float *)malloc(memsize); //Matriz creada
	float *B = (float *)malloc(memsize); //2da Matriz, que usaremos para calcular valores.
  float *d_a, *d_b; 

  hipMalloc(&d_a, memsize);
  hipMalloc(&d_b, memsize);
	//Primero vamos a inicializar la matriz A, dandole valores
	for(int i=0; i<N; ++i)
		for(int j=0;j<N;++j){
			if((i==0||i==N)&&(j==0||j==N))
				*(A+(i*N)+j)=150.0f;
			else
				*(A+(i*N)+j)=(float) (rand()%10);
	}
  
  hipMemcpy(d_a, A, memsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, B, memsize, hipMemcpyHostToDevice);

  dim3 block((N-2)/TAMBLOCK, (N-2)/TAMBLOCK);
  dim3 thread(TAMBLOCK, TAMBLOCK);

	//Forma 2 Jacobi, extremos = 150 y solo sumo por el interior.
  for(int k=0; k<LIMIT;++k){
		
    jacobi2d <<<block, thread>>> (d_a, d_b);

    float *aux=d_a;
		d_a=d_b;
		d_b=aux;
	}

  hipMemcpy(A, d_a, memsize, hipMemcpyDeviceToHost);  

	printf("\nValor del array A: \n");
	for(int i=0;i<N*N;++i){
		printf("%f ,",*(A+i));
		if(i!=0 && (i+1)%8==0)
			printf("\n");
	}
	
	//Este es el valor la matriz que contiene los valores anteriores.
	/*printf("\nValor del array B: \n");
	for(int i=0;i<ROW*COL;++i){
		printf("%f ,",*(B+i));
		if(i!=0 && (i+1)%5==0)
			printf("\n");
	}*/
}
