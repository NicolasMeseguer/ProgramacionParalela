#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 8 

__global__ void reduceVector(float *a, int thread){
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  for(int s=N/2; s>=1; s/=2){
    if(id<s)
      *(a+id) += *(a+id+s);
    __syncthreads();
  }
}

int main() {
  int memsize = sizeof(float)*N;
  float *a =(float *) malloc(memsize);
  float *d_a;

  for(int i=0;i<N;++i){
    *(a+i)=(float)(rand()%10);
    printf("%f ,", *(a+i));
  }
  printf("\n");

  hipMalloc(&d_a, memsize);
  hipMemcpy(d_a, a, memsize, hipMemcpyHostToDevice);

  int thread = (int) ceilf((double)N/2);
  reduceVector <<<1, thread>>> (d_a, thread);
    
  hipMemcpy(a, d_a, memsize, hipMemcpyDeviceToHost);
  printf("%f ,", *a);
  printf("\n\n");
  free(a);hipFree(d_a);
}
